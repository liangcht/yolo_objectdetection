#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <cfloat>
#include "caffe_cuda.h"

namespace {

template <typename scalar_t>
__global__ void smtl_cuda_forward_kernel(
    const int nthreads,
    const int* parent_data, const scalar_t* prob_data, const scalar_t* label,
    const int dim, const int spatial_dim,
    const bool has_ignore_label, const int ignore_label,
    scalar_t* loss_data, scalar_t* counts) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        // index == n * spatial_dim + s
        const int n = index / spatial_dim;
        const int s = index % spatial_dim;
        if (counts)
            counts[index] = 0;
        loss_data[index] = 0;
        int label_value = static_cast<int>(label[index]);
        if (has_ignore_label && label_value == ignore_label)
            continue;

        while (label_value >= 0) {
            loss_data[index] -= log(max(prob_data[n * dim + label_value * spatial_dim + s], scalar_t(FLT_MIN)));
            if (counts)
                counts[index]++;
            label_value = parent_data[label_value];
        }
    }
}

template <typename scalar_t>
__global__ void smtl_cuda_backward_kernel(
    const int nthreads,
    const int* parent_data, const int* group_offset_data, const int* group_size_data, const int* group_data,
    const scalar_t* label, const scalar_t* prob_data, scalar_t* bottom_diff,
    const int dim, const int spatial_dim,
    const bool has_ignore_label, const int ignore_label) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        // index == n * spatial_dim + s
        const int n = index / spatial_dim;
        const int s = index % spatial_dim;
        int label_value = static_cast<int>(label[index]);
        if (has_ignore_label && label_value == ignore_label)
            continue;
        while (label_value >= 0) {
            int g = group_data[label_value];
            int offset = group_offset_data[g];
            // TODO: Use dynamic parallelism for devices with 3.5 compute capability
            for (int c = 0; c < group_size_data[g]; ++c)
                bottom_diff[n * dim + (offset + c) * spatial_dim + s] = prob_data[n * dim + (offset + c) * spatial_dim + s];

            bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
            label_value = parent_data[label_value];
        }
    }
}

} // namespace

std::vector<at::Tensor> smtl_cuda_forward(
    at::Tensor prob, at::Tensor label,
    at::Tensor parent,
    int outer_num, int inner_num, int dim,
    bool has_ignore_label, int ignore_label, bool valid_normalization) {

  int nthreads = outer_num * inner_num;
  auto normalization = at::tensor(outer_num, prob.options());

  // Intermediate variables
  auto loss = at::empty_like(label);

  at::Tensor counts;
  AT_DISPATCH_FLOATING_TYPES(prob.type(), "smtl_cuda_forward", ([&] {
    scalar_t* counts_data = nullptr;
    if (valid_normalization) {
      counts = at::empty_like(label);
      counts_data = counts.data<scalar_t>();
    }
    smtl_cuda_forward_kernel<scalar_t><<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS>>>(
        nthreads,
        parent.data<int>(),
        prob.data<scalar_t>(),
        label.data<scalar_t>(),
        dim,
        inner_num,
        has_ignore_label, ignore_label,
        loss.data<scalar_t>(),
        counts_data);
    if (valid_normalization)
      normalization = counts.sum();
  }));

  return {loss.sum() / normalization, normalization};
}

std::vector<at::Tensor> smtl_cuda_backward(
    at::Tensor prob, at::Tensor label,
    at::Tensor parent, at::Tensor group_offset, at::Tensor group_size, at::Tensor group,
    int outer_num, int inner_num, int dim,
    bool has_ignore_label, int ignore_label) {

  int nthreads = outer_num * inner_num;

  auto diff = at::zeros_like(prob);

  AT_DISPATCH_FLOATING_TYPES(prob.type(), "smtl_cuda_backward", ([&] {
    smtl_cuda_backward_kernel<scalar_t><<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS>>>(
        nthreads,
        parent.data<int>(),
        group_offset.data<int>(),
        group_size.data<int>(),
        group.data<int>(),
        label.data<scalar_t>(),
        prob.data<scalar_t>(),
        diff.data<scalar_t>(),
        dim,
        inner_num,
        has_ignore_label, ignore_label);
  }));

  // Return un-normalzied
  return {diff};
}
